#include "hip/hip_runtime.h"
#include <stdint.h>
#include <hip/hip_fp16.h>
#include "ggml-cuda.h"

typedef uint16_t ggml_fp16_t;
static_assert(sizeof(__half) == sizeof(ggml_fp16_t), "wrong fp16 size");

#define QK4_0 32
typedef struct {
    float   d;              // delta
    uint8_t qs[QK4_0 / 2];  // nibbles / quants
} block_q4_0;
static_assert(sizeof(block_q4_0) == sizeof(float) + QK4_0 / 2, "wrong q4_0 block size/padding");

#define QK4_1 32
typedef struct {
    float   d;              // delta
    float   m;              // min
    uint8_t qs[QK4_1 / 2];  // nibbles / quants
} block_q4_1;
static_assert(sizeof(block_q4_1) == sizeof(float) * 2 + QK4_1 / 2, "wrong q4_1 block size/padding");

#define QK4_2 16
typedef struct {
    __half d;               // delta
    uint8_t qs[QK4_2 / 2];  // nibbles / quants
} block_q4_2;
static_assert(sizeof(block_q4_2) == sizeof(ggml_fp16_t) + QK4_2 / 2, "wrong q4_2 block size/padding");


static __global__ void dequantize_block_q4_0(const void * vx, float * y) {
    const block_q4_0 * x = (const block_q4_0 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;

    const uint8_t * pp = x[i].qs;

    for (int l = 0; l < QK4_0; l += 2) {
        const uint8_t vi = pp[l/2];

        const int8_t vi0 = vi & 0xf;
        const int8_t vi1 = vi >> 4;

        const float v0 = (vi0 - 8)*d;
        const float v1 = (vi1 - 8)*d;

        y[i*QK4_0 + l + 0] = v0;
        y[i*QK4_0 + l + 1] = v1;
    }
}

static __global__ void dequantize_block_q4_1(const void * vx, float * y) {
    const block_q4_1 * x = (const block_q4_1 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;
    const float m = x[i].m;

    const uint8_t * pp = x[i].qs;

    for (int l = 0; l < QK4_1; l += 2) {
        const uint8_t vi = pp[l/2];

        const int8_t vi0 = vi & 0xf;
        const int8_t vi1 = vi >> 4;

        const float v0 = vi0*d + m;
        const float v1 = vi1*d + m;

        y[i*QK4_1 + l + 0] = v0;
        y[i*QK4_1 + l + 1] = v1;
    }
}

static __global__ void dequantize_block_q4_2(const void * vx, float * y) {
    const block_q4_2 * x = (const block_q4_2 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;

    const uint8_t * pp = x[i].qs;

    for (int l = 0; l < QK4_2; l += 2) {
        const uint8_t vi = pp[l/2];

        const int8_t vi0 = vi & 0xf;
        const int8_t vi1 = vi >> 4;

        const float v0 = (vi0 - 8)*d;
        const float v1 = (vi1 - 8)*d;

        y[i*QK4_2 + l + 0] = v0;
        y[i*QK4_2 + l + 1] = v1;
    }
}

extern "C" {
    __host__ void dequantize_row_q4_0_cuda(const void * vx, float * y, int k, hipStream_t stream) {
        const int nb = k / QK4_0;
        dequantize_block_q4_0<<<nb, 1, 0, stream>>>(vx, y);
    }

    __host__ void dequantize_row_q4_1_cuda(const void * vx, float * y, int k, hipStream_t stream) {
        const int nb = k / QK4_1;
        dequantize_block_q4_1<<<nb, 1, 0, stream>>>(vx, y);
    }

    __host__ void dequantize_row_q4_2_cuda(const void * vx, float * y, int k, hipStream_t stream) {
        const int nb = k / QK4_2;
        dequantize_block_q4_2<<<nb, 1, 0, stream>>>(vx, y);
    }
}
